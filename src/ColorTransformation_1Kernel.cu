#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"

__global__ void ColorTransformation(unsigned char* input, unsigned char* outputColorInvariance, unsigned char* outputGrayscale, unsigned char* outputU, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int pixelIndex = y * width + x;
        int rgbIndex = pixelIndex * 3;

        float R = input[rgbIndex];
        float G = input[rgbIndex + 1];
        float B = input[rgbIndex + 2];

        float r_prime = atanf(R / fmaxf(G, B));
        float g_prime = atanf(G / fmaxf(R, B));
        float b_prime = atanf(B / fmaxf(R, G));

        outputColorInvariance[rgbIndex] = static_cast<unsigned char>((r_prime / (M_PI / 2)) * 255);
        outputColorInvariance[rgbIndex + 1] = static_cast<unsigned char>((g_prime / (M_PI / 2)) * 255);
        outputColorInvariance[rgbIndex + 2] = static_cast<unsigned char>((b_prime / (M_PI / 2)) * 255);

        unsigned char grayValue = static_cast<unsigned char>(0.21f * outputColorInvariance[rgbIndex] + 0.71f * outputColorInvariance[rgbIndex + 1] + 0.07f * outputColorInvariance[rgbIndex + 2]);
        outputGrayscale[pixelIndex] = grayValue;

        outputU[pixelIndex] = static_cast<unsigned char>(128 + (-0.147 * R - 0.289 * G + 0.436 * B));
    }
}


int main(int argc, char* argv[]) {
    if (argc < 5) {
        std::cerr << "Usage: " << argv[0] << " <inputImagePath> <colorInvarianceOutputPath> <grayscaleOutputPath> <UComponentOutputPath>" << std::endl;
        return -1;
    }

    const char* inputImagePath = argv[1];
    const char* colorInvarianceOutputPath = argv[2];
    const char* grayscaleOutputPath = argv[3];
    const char* UComponentOutputPath = argv[4];

    int width, height, channels;
    unsigned char* inputImage = stbi_load(inputImagePath, &width, &height, &channels, 0);

    if (inputImage == NULL) {
        std::cerr << "Error loading image: " << inputImagePath << std::endl;
        return -1;
    }

    if (channels < 3) {
        std::cerr << "Error: Image must have at least 3 channels (RGB)" << std::endl;
        stbi_image_free(inputImage);
        return -1;
    }

    int imageSize = width * height * channels;

    unsigned char *d_inputImage, *d_colorInvarianceImage, *d_grayscaleImage, *d_UComponentImage;
    hipMalloc((void**)&d_inputImage, imageSize * sizeof(unsigned char));
    hipMalloc((void**)&d_colorInvarianceImage, imageSize * sizeof(unsigned char)); // 3 channels
    hipMalloc((void**)&d_grayscaleImage, width * height * sizeof(unsigned char)); // Single channel
    hipMalloc((void**)&d_UComponentImage, width * height * sizeof(unsigned char)); // U component

    hipMemcpy(d_inputImage, inputImage, imageSize, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Launch the combined kernel
    ColorTransformation<<<gridDim, blockDim>>>(d_inputImage, d_colorInvarianceImage, d_grayscaleImage, d_UComponentImage, width, height);

    // Allocate memory for output images
    unsigned char* colorInvarianceImage = new unsigned char[imageSize];
    unsigned char* grayscaleImage = new unsigned char[width * height];
    unsigned char* UComponentImage = new unsigned char[width * height];

    // Copy the converted images back to host
    hipMemcpy(colorInvarianceImage, d_colorInvarianceImage, imageSize, hipMemcpyDeviceToHost);
    hipMemcpy(grayscaleImage, d_grayscaleImage, width * height, hipMemcpyDeviceToHost);
    hipMemcpy(UComponentImage, d_UComponentImage, width * height, hipMemcpyDeviceToHost);

    // Save the output images
    stbi_write_jpg(colorInvarianceOutputPath, width, height, 3, colorInvarianceImage, 100);
    stbi_write_jpg(grayscaleOutputPath, width, height, 1, grayscaleImage, 100);
    stbi_write_jpg(UComponentOutputPath, width, height, 1, UComponentImage, 100);

    // Cleanup
    stbi_image_free(inputImage);
    delete[] colorInvarianceImage;
    delete[] grayscaleImage;
    delete[] UComponentImage;
    hipFree(d_inputImage);
    hipFree(d_colorInvarianceImage);
    hipFree(d_grayscaleImage);
    hipFree(d_UComponentImage);

    return 0;
}

