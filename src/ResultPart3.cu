
#include <hip/hip_runtime.h>
__global__ void colorConversionKernel(float* result, float* image_double, float* smoothmask, float* ratio_red, float* ratio_green, float* ratio_blue, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x < width && y < height) {
        int index = y * width + x;

        float smooth_val = 1 - smoothmask[index];

        result[index] = (ratio_red[index] + 1) / (smooth_val * ratio_red[index] + 1) * image_double[index];
        result[index + width * height] = (ratio_green[index] + 1) / (smooth_val * ratio_green[index] + 1) * image_double[index + width * height];
        result[index + 2 * width * height] = (ratio_blue[index] + 1) / (smooth_val * ratio_blue[index] + 1) * image_double[index + 2 * width * height];
    }
}

void convertMatlabToCUDA(float* result, float* image_double, float* smoothmask, float* ratio_red, float* ratio_green, float* ratio_blue, int width, int height) {
    int size = width * height * sizeof(float);
    float *d_result, *d_image_double, *d_smoothmask, *d_ratio_red, *d_ratio_green, *d_ratio_blue;

    hipMalloc(&d_result, 3 * size);
    hipMalloc(&d_image_double, 3 * size);
    hipMalloc(&d_smoothmask, size);
    hipMalloc(&d_ratio_red, size);
    hipMalloc(&d_ratio_green, size);
    hipMalloc(&d_ratio_blue, size);

    hipMemcpy(d_image_double, image_double, 3 * size, hipMemcpyHostToDevice);
    hipMemcpy(d_smoothmask, smoothmask, size, hipMemcpyHostToDevice);
    hipMemcpy(d_ratio_red, ratio_red, size, hipMemcpyHostToDevice);
    hipMemcpy(d_ratio_green, ratio_green, size, hipMemcpyHostToDevice);
    hipMemcpy(d_ratio_blue, ratio_blue, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((width + 15) / 16, (height + 15) / 16);
    colorConversionKernel<<<numBlocks, threadsPerBlock>>>(d_result, d_image_double, d_smoothmask, d_ratio_red, d_ratio_green, d_ratio_blue, width, height);

    hipMemcpy(result, d_result, 3 * size, hipMemcpyDeviceToHost);

    hipFree(d_result);
    hipFree(d_image_double);
    hipFree(d_smoothmask);
    hipFree(d_ratio_red);
    hipFree(d_ratio_green);
    hipFree(d_ratio_blue);
}
