#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include "shadowRemoval.cu"
#include "ConvolutionKernal.cu"
#include "Erosion.cu"
#include <wb.h>


#define CUDA_CHECK(ans)                                                   \
  { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line,
                      bool abort = true) {
  if (code != hipSuccess) {
    fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
            file, line);
    if (abort)
      exit(code);
  }
}

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <inputImagePath> <colorInvarianceOutputPath> <grayscaleOutputPath> <UComponentOutputPath>" << std::endl;
        return -1;
    }

    // For calculating proccesses
    hipEvent_t astartEvent, astopEvent;
    float aelapsedTime;
    hipEventCreate(&astartEvent);
    hipEventCreate(&astopEvent);
    
    // for total time
    hipEvent_t atotalStartEvent, atotalStopEvent;
    float atotalElapsedTime;
    hipEventCreate(&atotalStartEvent);
    hipEventCreate(&atotalStopEvent);

    
    //if (argc < 5) {
    //    std::cerr << "Usage: " << argv[0] << " <inputImagePath> <colorInvarianceOutputPath> <grayscaleOutputPath> <UComponentOutputPath>" << std::endl;
    //    return -1;
    //}



    const char* inputImagePath = argv[1];
    const char* colorInvarianceOutputPath = argv[2];
    const char* grayscaleOutputPath = argv[3];
    const char* UComponentOutputPath = argv[4];
    const char* ConvoOutputPath = argv[5];
    const char* ErodedLightOutputPath = argv[6];
    const char* ErodedShadowOutputPath  = argv[7];
    const char* FinalOutputPath = argv[8];

    int width, height, channels;
    unsigned char* inputImage = stbi_load(inputImagePath, &width, &height, &channels, 0);
     int Mask_Width;

    if (inputImage == NULL) {
        std::cerr << "Error loading image: " << inputImagePath << std::endl;
        return -1;
    }

    if (channels < 3) {
        std::cerr << "Error: Image must have at least 3 channels (RGB)" << std::endl;
        stbi_image_free(inputImage);
        return -1;
    }

    hipEventRecord(atotalStartEvent, 0);

    int imageSize = width * height * channels;
    int grayscaleSize = width * height;

    unsigned char *d_inputImage, *d_colorInvarianceImage, *d_grayscaleImage, *d_UComponentImage, *d_GreyScaleMask, *d_YUVMask, *d_ConvoOutput;
    unsigned char *d_greyscalethreshold, *d_yuvthreshold;

    wbTime_start(GPU, "Copying input memory to the GPU.");

    hipMalloc((void**)&d_inputImage, imageSize * sizeof(unsigned char));
    hipMalloc((void**)&d_colorInvarianceImage, imageSize * sizeof(unsigned char)); // 3 channels
    hipMalloc((void**)&d_grayscaleImage, grayscaleSize * sizeof(unsigned char)); // Single channel
    hipMalloc((void**)&d_UComponentImage, grayscaleSize * sizeof(unsigned char)); // U component
    //hipMalloc((void**)&d_UComponentImage, grayscaleSize * sizeof(unsigned char)); // U component
    hipMalloc((void**)&d_ConvoOutput, grayscaleSize * sizeof(unsigned char)); // U component

    hipMalloc((void**)&d_GreyScaleMask, grayscaleSize * sizeof(unsigned char)); // greymask 
    hipMalloc((void**)&d_YUVMask, grayscaleSize * sizeof(unsigned char)); // YUV Mask

    hipMemcpy(d_inputImage, inputImage, imageSize, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);
    hipEventRecord(astartEvent, 0);
    // Launch the combined kernel
    ColorTransformation<<<gridDim, blockDim>>>(d_inputImage, d_colorInvarianceImage, d_grayscaleImage, d_UComponentImage, width, height);

    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    printf("\n");
    printf("Total time for Color transforms Proccess 1 (ms) %f \n",aelapsedTime);
    printf("\n");


    hipEventRecord(astartEvent, 0);
    // Allocate memory for the histogram
    unsigned int* d_histogram;
    hipMalloc((void**)&d_histogram, NUM_BINS * sizeof(unsigned int));
    hipMemset(d_histogram, 0, NUM_BINS * sizeof(unsigned int));

    // Configure kernel launch parameters
    int threadsPerBlock = 256;  // This is a typical choice; adjust based on GPU
    int numBlocks = (width * height + threadsPerBlock - 1) / threadsPerBlock;

    // Launch histogram kernel for grayscale image
    computeHistogram<<<numBlocks, threadsPerBlock, NUM_BINS * sizeof(unsigned int)>>>(d_grayscaleImage, d_histogram, width * height);
    //calculateOtsuThreshold <<<numBlocks, threadsPerBlock, NUM_BINS * sizeof(unsigned int)>>>(d_histogram,imageSize,d_greyscalethreshold);
    // Launch histogram kernel for yuv image
    computeHistogram<<<numBlocks, threadsPerBlock, NUM_BINS * sizeof(unsigned int)>>>(d_UComponentImage, d_histogram, width * height);

    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    printf("\n");
    printf("Total time for Proccess 2 Otsu (ms) %f \n",aelapsedTime);
    printf("\n");

    // for proccess 3 convolution
    hipEventRecord(astartEvent, 0);
    Mask_Width =  11;
    convolution_basic_kernel<<<gridDim, blockDim>>>(d_UComponentImage, d_YUVMask, d_ConvoOutput, Mask_Width, width, height) ;

    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    printf("\n");
    printf("Total time for Proccess 3 convolutions (ms) %f \n",aelapsedTime);
    printf("\n");

     // for proccess 4
    hipEventRecord(astartEvent, 0);
// Configure Erosion Kernel
    // Allocate memory for input image (We will take the gray mask)
        // Do this for sahdow and light mask. 1-mask is the light mask.
    unsigned char* d_erodedMaskShadow;
    unsigned char* d_erodedMaskLight;
    // Allocate input 
        // Cuda host to device copy of the input mask.
    // Allocate output
    hipMalloc((void**)&d_erodedMaskShadow, grayscaleSize * sizeof(unsigned char));
    hipMalloc((void**)&d_erodedMaskLight, grayscaleSize * sizeof(unsigned char));
    dim3 erodeBlock(32, 32);
    dim3 erodeGrid(ceil((float)width/erodeBlock.x), ceil((float)height / erodeBlock.y));

    Erosion<<<erodeGrid, erodeBlock>>>(d_GreyScaleMask, d_erodedMaskShadow, width, height, 2);
    //finish erases

    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    printf("\n");
    printf("Total time for Proccess 4 errosion (ms) %f \n",aelapsedTime);
    printf("\n");


     // for proccess 5
    hipEventRecord(astartEvent, 0);


     hipEventRecord(astopEvent, 0);
    hipEventSynchronize(astopEvent);
    hipEventElapsedTime(&aelapsedTime, astartEvent, astopEvent);
    printf("\n");
    printf("Total time for Proccess 5 result (ms) %f \n",aelapsedTime);
    printf("\n");


    hipEventRecord(astopEvent, 0);
    hipEventSynchronize(atotalStopEvent);
    hipEventElapsedTime(&atotalElapsedTime, atotalStartEvent, atotalStopEvent);
    printf("\n");
    printf("Total compute time of function after proccess 5 commits(ms) %f \n",aelapsedTime);
    printf("\n");


// Copy histogram back to host
unsigned int* histogram = new unsigned int[NUM_BINS];
hipMemcpy(histogram, d_histogram, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    //Printing histogram values
    printf("Histogram values:\n");
    for (int i = 0; i < NUM_BINS; i++) {
        printf("%d: %u, ", i, histogram[i]);
        if ((i + 1) % 16 == 0) printf("\n");
    }
    printf("\n");
    // Allocate memory for output images
    unsigned char* colorInvarianceImage = new unsigned char[imageSize];
    unsigned char* grayscaleImage = new unsigned char[grayscaleSize];
    unsigned char* UComponentImage = new unsigned char[grayscaleSize];
    unsigned char* ConvoOutput = new unsigned char[grayscaleSize];
    unsigned char* ErodedLight = new unsigned char[grayscaleSize];
    unsigned char* ErodedShadow = new unsigned char[grayscaleSize];
    unsigned char* Final = new unsigned char[imageSize];




    // Copy the converted images back to host
    hipMemcpy(colorInvarianceImage, d_colorInvarianceImage, imageSize, hipMemcpyDeviceToHost);
    hipMemcpy(grayscaleImage, d_grayscaleImage, grayscaleSize, hipMemcpyDeviceToHost);
    hipMemcpy(UComponentImage, d_UComponentImage, grayscaleSize, hipMemcpyDeviceToHost);
    hipMemcpy(ConvoOutput, d_ConvoOutput, grayscaleSize, hipMemcpyDeviceToHost);

    hipMemcpy(ErodedLight, d_erodedMaskShadow, grayscaleSize, hipMemcpyDeviceToHost);
    hipMemcpy(ErodedShadow, d_erodedMaskLight, grayscaleSize, hipMemcpyDeviceToHost);
    //hipMemcpy(Final, d_ConvoOutput, imageSize, hipMemcpyDeviceToHost);


    // Save the output images
    stbi_write_jpg(colorInvarianceOutputPath, width, height, 3, colorInvarianceImage, 100);
    stbi_write_jpg(grayscaleOutputPath, width, height, 1, grayscaleImage, 100);
    stbi_write_jpg(UComponentOutputPath, width, height, 1, UComponentImage, 100);
    stbi_write_jpg(ConvoOutputPath, width, height, 1, ConvoOutput, 100);

    stbi_write_jpg(ErodedLightOutputPath, width, height, 1, ErodedLight, 100);
    stbi_write_jpg(ErodedShadowOutputPath, width, height, 1, ErodedShadow, 100);
    //stbi_write_jpg(FinalOutputPath, width, height, 1, Final, 100);


    // Cleanup
    stbi_image_free(inputImage);
    delete[] colorInvarianceImage;
    delete[] grayscaleImage;
    delete[] UComponentImage;
    delete[] d_ConvoOutput;
    delete[] histogram;
    delete[] d_GreyScaleMask;
    delete[] d_YUVMask;
    delete[] d_erodedMaskLight;
    delete[] d_erodedMaskLight;

    hipFree(d_inputImage);
    hipFree(d_colorInvarianceImage);
    hipFree(d_grayscaleImage);
    hipFree(d_UComponentImage);
    hipFree(d_ConvoOutput);
    hipFree(d_histogram);
    hipFree(d_GreyScaleMask);
    hipFree(d_YUVMask);
    hipFree(d_erodedMaskLight);
    hipFree(d_erodedMaskLight);


    return 0;
}

