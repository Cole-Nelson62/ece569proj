#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib>
#include <cstdio>
#include <cmath>
#include <hip/hip_runtime.h>
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image.h"
#define STB_IMAGE_WRITE_IMPLEMENTATION
#include "stb_image_write.h"
#include "shadowRemoval.cu"
#include "ConvolutionKernal.cu"
#include "Erosion.cu"

int main(int argc, char* argv[]) {
    if (argc < 2) {
        std::cerr << "Usage: " << argv[0] << " <inputImagePath> <colorInvarianceOutputPath> <grayscaleOutputPath> <UComponentOutputPath>" << std::endl;
        return -1;
    }
    //if (argc < 5) {
    //    std::cerr << "Usage: " << argv[0] << " <inputImagePath> <colorInvarianceOutputPath> <grayscaleOutputPath> <UComponentOutputPath>" << std::endl;
    //    return -1;
    //}

    const char* inputImagePath = argv[1];
    const char* colorInvarianceOutputPath = argv[2];
    const char* grayscaleOutputPath = argv[3];
    const char* UComponentOutputPath = argv[4];

    int width, height, channels;
    unsigned char* inputImage = stbi_load(inputImagePath, &width, &height, &channels, 0);

    if (inputImage == NULL) {
        std::cerr << "Error loading image: " << inputImagePath << std::endl;
        return -1;
    }

    if (channels < 3) {
        std::cerr << "Error: Image must have at least 3 channels (RGB)" << std::endl;
        stbi_image_free(inputImage);
        return -1;
    }

    int imageSize = width * height * channels;
    int grayscaleSize = width * height;

    unsigned char *d_inputImage, *d_colorInvarianceImage, *d_grayscaleImage, *d_UComponentImage;
    hipMalloc((void**)&d_inputImage, imageSize * sizeof(unsigned char));
    hipMalloc((void**)&d_colorInvarianceImage, imageSize * sizeof(unsigned char)); // 3 channels
    hipMalloc((void**)&d_grayscaleImage, grayscaleSize * sizeof(unsigned char)); // Single channel
    hipMalloc((void**)&d_UComponentImage, grayscaleSize * sizeof(unsigned char)); // U component

    hipMemcpy(d_inputImage, inputImage, imageSize, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, (height + blockDim.y - 1) / blockDim.y);

    // Launch the combined kernel
    ColorTransformation<<<gridDim, blockDim>>>(d_inputImage, d_colorInvarianceImage, d_grayscaleImage, d_UComponentImage, width, height);

    // Allocate memory for the histogram
unsigned int* d_histogram;
hipMalloc((void**)&d_histogram, NUM_BINS * sizeof(unsigned int));
hipMemset(d_histogram, 0, NUM_BINS * sizeof(unsigned int));

// Configure kernel launch parameters
int threadsPerBlock = 256;  // This is a typical choice; adjust based on GPU
int numBlocks = (width * height + threadsPerBlock - 1) / threadsPerBlock;

// Launch histogram kernel for grayscale image
computeHistogram<<<numBlocks, threadsPerBlock, NUM_BINS * sizeof(unsigned int)>>>(d_grayscaleImage, d_histogram, width * height);

// Configure Erosion Kernel
    // Allocate memory for input image (We will take the gray mask)
        // Do this for sahdow and light mask. 1-mask is the light mask.
    unsigned char* d_erodedMaskShadow;
    unsigned char* d_erodedMaskLight;
    // Allocate input 
        // Cuda host to device copy of the input mask.
    // Allocate output
    hipMalloc((void**)&d_erodedMaskShadow, grayscaleSize * sizeof(unsigned char));
    hipMalloc((void**)&d_erodedMaskLight, grayscaleSize * sizeof(unsigned char));
    dim3 erodeBlock(32, 32);
    dim3 erodeGrid(ceil((float)width/erodeBlock.x), ceil((float)height / erodeBlock.y));

    Erosion<<<erodeGrid, erodeBlock>>>(MASK, d_erodedMaskShadow, width, height, 2);
    //finish erases


// Copy histogram back to host
unsigned int* histogram = new unsigned int[NUM_BINS];
hipMemcpy(histogram, d_histogram, NUM_BINS * sizeof(unsigned int), hipMemcpyDeviceToHost);
    //Printing histogram values
    printf("Histogram values:\n");
    for (int i = 0; i < NUM_BINS; i++) {
        printf("%d: %u, ", i, histogram[i]);
        if ((i + 1) % 16 == 0) printf("\n");
    }
    printf("\n");
    // Allocate memory for output images
    unsigned char* colorInvarianceImage = new unsigned char[imageSize];
    unsigned char* grayscaleImage = new unsigned char[grayscaleSize];
    unsigned char* UComponentImage = new unsigned char[grayscaleSize];

    // Copy the converted images back to host
    hipMemcpy(colorInvarianceImage, d_colorInvarianceImage, imageSize, hipMemcpyDeviceToHost);
    hipMemcpy(grayscaleImage, d_grayscaleImage, grayscaleSize, hipMemcpyDeviceToHost);
    hipMemcpy(UComponentImage, d_UComponentImage, grayscaleSize, hipMemcpyDeviceToHost);

    // Save the output images
    stbi_write_jpg(colorInvarianceOutputPath, width, height, 3, colorInvarianceImage, 100);
    stbi_write_jpg(grayscaleOutputPath, width, height, 1, grayscaleImage, 100);
    stbi_write_jpg(UComponentOutputPath, width, height, 1, UComponentImage, 100);

    // Cleanup
    stbi_image_free(inputImage);
    delete[] colorInvarianceImage;
    delete[] grayscaleImage;
    delete[] UComponentImage;
    delete[] histogram;
    hipFree(d_inputImage);
    hipFree(d_colorInvarianceImage);
    hipFree(d_grayscaleImage);
    hipFree(d_UComponentImage);
    hipFree(d_histogram);


    return 0;
}

